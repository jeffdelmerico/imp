#include "hip/hip_runtime.h"
#ifndef IMP_CU_MEDIAN3X3_IMPL_CU
#define IMP_CU_MEDIAN3X3_IMPL_CU

#include <imp/cuimgproc/cu_image_filter.cuh>

#include <cstdint>
#include <cfloat>
#include <hip/hip_runtime.h>

#include <imp/core/types.hpp>
#include <imp/core/roi.hpp>
#include <imp/cucore/cu_image_gpu.cuh>
#include <imp/cucore/cu_utils.hpp>
#include <imp/cucore/cu_texture.cuh>



namespace imp {
namespace cu {

//-----------------------------------------------------------------------------
template<typename Pixel>
__global__ void  k_median3x3(Pixel* dst, const size_type stride,
                             const std::uint32_t xoff, const std::uint32_t yoff,
                             const std::uint32_t width, const std::uint32_t height,
                             Texture2D src_tex)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  const size_type out_idx = y*stride+x;

  if(x>=0 && y>= 0 && x<width && y<height)
  {
    x += xoff;
    y += yoff;

    // shared mem coords
    const int tx = threadIdx.x+1;
    const int ty = threadIdx.y+1;
    // we have a 3x3 kernel, so our width of the shared memory (shp) is blockDim.x + 2!
    const int shp = blockDim.x + 2;
    const int shc = (threadIdx.y+1) * shp + (threadIdx.x+1);
    extern __shared__ float sh_in[];

    // Load input 3x3 block into shared memory
    // Note: the FLT_MAX prevents us from overemphasizing the border pixels if they are outliers!
    {
      // for each thread: copy the data of the current input position to shared mem
      Pixel texel;
      src_tex.fetch(texel, x, y);
      sh_in[shc] = texel;

      /////////////////////////////////////////////////////////////////////////////
      // boundary conditions
      /////////////////////////////////////////////////////////////////////////////
      if (x == 0) // at left image border
      {
        if (y == 0)
          sh_in[shc-shp-1] = FLT_MAX; // left-upper corner (image)
        else if (ty == 1)
        {
          // left-upper corner (block)
          src_tex.fetch(texel, x, y-1);
          sh_in[shc-shp-1] = texel;
        }

        sh_in[shc-1] = sh_in[shc];     // left border (image)

        if (y == height-1)
          sh_in[shc+shp-1] = FLT_MAX; // left-lower corner (image)
        else if (ty == blockDim.y)
        {
          src_tex.fetch(texel, x, y+1);
          sh_in[shc+shp-1] = texel; // left-lower corner (block)
        }
      }
      else if (tx == 1) // at left block border (inside image w.r.t x)
      {
        if (y == 0)
        {
          src_tex.fetch(texel, x-1, y);
          sh_in[shc-shp-1] = texel; // left-upper corner (block, outside)
        }
        else if (ty == 1)
        {
          src_tex.fetch(texel, x-1, y-1);
          sh_in[shc-shp-1] = texel; // left-upper corner (block, inside)
        }

        src_tex.fetch(texel, x-1, y);
        sh_in[shc-1] = texel; // left border (block)

        if (y == height-1)
        {
          src_tex.fetch(texel, x-1, y);
          sh_in[shc+shp-1] = texel; // left-lower corner (block, outside)
        }
        else if (ty == blockDim.y)
        {
          src_tex.fetch(texel, x-1, y+1);
          sh_in[shc+shp-1] = texel; // left-lower corner (block, inside)
        }
      }


      if (x == width-1) // at right image border
      {
        if (y == 0)
          sh_in[shc-shp+1] = FLT_MAX; // right-upper corner (image)
        else if (ty == 1)
        {
          src_tex.fetch(texel, x, y-1);
          sh_in[shc-shp+1] = texel; // right-upper corner (block)
        }

        sh_in[shc+1] = sh_in[shc]; // right border (image)

        if (y == height-1)
          sh_in[shc+shp+1] = FLT_MAX; // right-lower corner (image)
        else if (ty == blockDim.y)
        {
          src_tex.fetch(texel, x, y+1);
          sh_in[shc+shp+1] = texel; // right-lower corner (block)
        }
      }
      else if (tx == blockDim.x) // at right block border (inside image w.r.t x)
      {
        if (y == 0)
        {
          src_tex.fetch(texel, x+1, y);
          sh_in[shc-shp+1] = texel; // right-upper corner (block, outside)
        }
        else if (ty == 1)
        {
          src_tex.fetch(texel, x+1, y-1);
          sh_in[shc-shp+1] = texel; // right-upper corner (block, inside)
        }

        src_tex.fetch(texel, x+1, y);
        sh_in[shc+1] = texel; // right border (block)

        if (y == height-1)
        {
          src_tex.fetch(texel, x+1, y);
          sh_in[shc+shp+1] = texel; // right-lower corner (block, outside)
        }
        else if (ty == blockDim.y)
        {
          src_tex.fetch(texel, x+1, y+1);
          sh_in[shc+shp+1] = texel; // right-lower corner (block, inside)
        }
      }

      if (y == 0)
        sh_in[shc-shp] = sh_in[shc]; // upper border (image)
      else if (ty == 1)
      {
        src_tex.fetch(texel, x, y-1);
        sh_in[shc-shp] = texel; // upper border (block)
      }

      if (y == height-1)
        sh_in[shc+shp] = sh_in[shc]; // lower border (image)
      else if (ty == blockDim.y)
      {
        src_tex.fetch(texel, x, y+1);
        sh_in[shc+shp] = texel; // lower border (block)
      }

      __syncthreads();
    }

    // in a sequence of nine elements, we have to remove four times the maximum from the sequence and need
    // a fifth calculated maximum which is the median!

    float maximum;
    {
      float vals[8];

      // first 'loop'
      vals[0] = fmin(sh_in[shc-shp-1], sh_in[shc-shp]);
      maximum = fmax(sh_in[shc-shp-1], sh_in[shc-shp]);
      vals[1] = fmin(maximum, sh_in[shc-shp+1]);
      maximum = fmax(maximum, sh_in[shc-shp+1]);
      vals[2] = fmin(maximum, sh_in[shc-1]);
      maximum = fmax(maximum, sh_in[shc-1]);
      vals[3] = fmin(maximum, sh_in[shc]);
      maximum = fmax(maximum, sh_in[shc]);
      vals[4] = fmin(maximum, sh_in[shc+1]);
      maximum = fmax(maximum, sh_in[shc+1]);
      vals[5] = fmin(maximum, sh_in[shc+shp-1]);
      maximum = fmax(maximum, sh_in[shc+shp-1]);
      vals[6] = fmin(maximum, sh_in[shc+shp]);
      maximum = fmax(maximum, sh_in[shc+shp]);
      vals[7] = fmin(maximum, sh_in[shc+shp+1]);
      maximum = fmax(maximum, sh_in[shc+shp+1]);

      // second 'loop'
      maximum = fmax(vals[0], vals[1]);
      vals[0] = fmin(vals[0], vals[1]);
      vals[1] = maximum;
      maximum = fmax(vals[1], vals[2]);
      vals[1] = fmin(vals[1], vals[2]);
      vals[2] = maximum;
      maximum = fmax(vals[2], vals[3]);
      vals[2] = fmin(vals[2], vals[3]);
      vals[3] = maximum;
      maximum = fmax(vals[3], vals[4]);
      vals[3] = fmin(vals[3], vals[4]);
      vals[4] = maximum;
      maximum = fmax(vals[4], vals[5]);
      vals[4] = fmin(vals[4], vals[5]);
      vals[5] = maximum;
      maximum = fmax(vals[5], vals[6]);
      vals[5] = fmin(vals[5], vals[6]);
      vals[6] = fmin(maximum, vals[7]);

      // third 'loop'
      maximum = fmax(vals[0], vals[1]);
      vals[0] = fmin(vals[0], vals[1]);
      vals[1] = maximum;
      maximum = fmax(vals[1], vals[2]);
      vals[1] = fmin(vals[1], vals[2]);
      vals[2] = maximum;
      maximum = fmax(vals[2], vals[3]);
      vals[2] = fmin(vals[2], vals[3]);
      vals[3] = maximum;
      maximum = fmax(vals[3], vals[4]);
      vals[3] = fmin(vals[3], vals[4]);
      vals[4] = maximum;
      maximum = fmax(vals[4], vals[5]);
      vals[4] = fmin(vals[4], vals[5]);
      vals[5] = fmin(maximum, vals[6]);

      // 4th 'loop'
      maximum = fmax(vals[0], vals[1]);
      vals[0] = fmin(vals[0], vals[1]);
      vals[1] = maximum;
      maximum = fmax(vals[1], vals[2]);
      vals[1] = fmin(vals[1], vals[2]);
      vals[2] = maximum;
      maximum = fmax(vals[2], vals[3]);
      vals[2] = fmin(vals[2], vals[3]);
      vals[3] = maximum;
      maximum = fmax(vals[3], vals[4]);
      vals[3] = fmin(vals[3], vals[4]);
      vals[4] = fmin(maximum, vals[5]);

      // 5th 'loop'
      maximum = fmax(vals[0], vals[1]);
      maximum = fmax(maximum, vals[2]);
      maximum = fmax(maximum, vals[3]);
      maximum = fmax(maximum, vals[4]);
    }
    dst[out_idx] = maximum;
  }
}

//-----------------------------------------------------------------------------
template<typename Pixel, imp::PixelType pixel_type>
void filterMedian3x3(ImageGpu<Pixel, pixel_type>* dst,
                     ImageGpu<Pixel, pixel_type>* src)
{
  std::unique_ptr<Texture2D> src_tex =
      src->genTexture(false,(src->bitDepth()<32) ? hipFilterModePoint
                                                 : hipFilterModeLinear);

  std::uint16_t block_size = 16;
  Fragmentation<16,16> frag(src->roi());
  size_type shared_size = (block_size+2)*(block_size+2)*sizeof(float);

  Roi2u roi = src->roi();
  dst->setRoi(roi);

  k_median3x3
      <<<
        frag.dimGrid, frag.dimBlock, shared_size
      >>> (
          dst->data(roi.x(), roi.y()), dst->stride(),
          roi.x(), roi.y(), roi.width(), roi.height(), *src_tex);

  IMP_CUDA_CHECK();
}

//==============================================================================
//
// template instantiations for all our image types
//

template void filterMedian3x3(ImageGpu8uC1* dst, ImageGpu8uC1* src);
template void filterMedian3x3(ImageGpu8uC2* dst, ImageGpu8uC2* src);
template void filterMedian3x3(ImageGpu8uC4* dst, ImageGpu8uC4* src);

template void filterMedian3x3(ImageGpu16uC1* dst, ImageGpu16uC1* src);
template void filterMedian3x3(ImageGpu16uC2* dst, ImageGpu16uC2* src);
template void filterMedian3x3(ImageGpu16uC4* dst, ImageGpu16uC4* src);

template void filterMedian3x3(ImageGpu32sC1* dst, ImageGpu32sC1* src);
template void filterMedian3x3(ImageGpu32sC2* dst, ImageGpu32sC2* src);
template void filterMedian3x3(ImageGpu32sC4* dst, ImageGpu32sC4* src);

template void filterMedian3x3(ImageGpu32fC1* dst, ImageGpu32fC1* src);
template void filterMedian3x3(ImageGpu32fC2* dst, ImageGpu32fC2* src);
template void filterMedian3x3(ImageGpu32fC4* dst, ImageGpu32fC4* src);


} // namespace cu
} // namespace imp



#endif // IMP_CU_MEDIAN3X3_IMPL_CU
