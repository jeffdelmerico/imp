#include "hip/hip_runtime.h"
#include <imp/cudepth/solver_epipolar_stereo_precond_huber_l1.cuh>

#include <hip/hip_runtime.h>

#include <glog/logging.h>

#include <imp/cudepth/variational_stereo_parameters.hpp>
#include <imp/cucore/cu_image_gpu.cuh>
#include <imp/cuimgproc/cu_image_filter.cuh>
#include <imp/cuimgproc/cu_image_transform.cuh>
#include <imp/cucore/cu_utils.hpp>
#include <imp/cucore/cu_texture.cuh>
#include <imp/cucore/cu_math.cuh>
#include <imp/cucore/cu_k_setvalue.cuh>
#include <imp/cuimgproc/edge_detectors.cuh>

#include "cu_k_warped_gradients.cuh"
//#include "cu_k_stereo_ctf_warping_level_precond_huber_l1.cuh"
#include "cu_k_solver_epipolar_stereo_precond_huber_l1.cuh"
#include "cu_k_stereo_ctf_warping_level_precond_huber_l1_weighted.cuh"

//#include "k_epipolar_stereo_precond_huber_l1.cu"

#define USE_EDGES 0

namespace imp {
namespace cu {

//------------------------------------------------------------------------------
SolverEpipolarStereoPrecondHuberL1::~SolverEpipolarStereoPrecondHuberL1()
{
  // thanks to smart pointers
}

//------------------------------------------------------------------------------
SolverEpipolarStereoPrecondHuberL1::SolverEpipolarStereoPrecondHuberL1(
    const std::shared_ptr<Parameters>& params, imp::Size2u size, size_type level,
    const std::vector<cu::PinholeCamera>& cams,
    const cu::Matrix3f& F,
    const std::vector<Vec32fC2>& epipoles,
    const cu::SE3<float>& T_mov_fix,
    const imp::cu::ImageGpu32fC1& depth_proposal,
    const imp::cu::ImageGpu32fC1& depth_proposal_sigma2)
  : SolverStereoAbstract(params, size, level)
  , F_(F)
  , epipoles_(epipoles)
  , T_mov_fix_(T_mov_fix)

{
  u_.reset(new DisparityImage(size));
  u_prev_.reset(new Image(size));
  u0_.reset(new Image(size));
  pu_.reset(new VectorImage(size));
  q_.reset(new Image(size));
  ix_.reset(new Image(size));
  it_.reset(new Image(size));
  xi_.reset(new Image(size));
  g_.reset(new Image(size));

  mu_.reset(new DisparityImage(size));
  sigma2_.reset(new DisparityImage(size));
  pt_mu_.reset(new imp::cu::ImageGpu32fC2(size));
  epi_vec_.reset(new imp::cu::ImageGpu32fC2(size));

  float scale_factor = std::pow(params->ctf.scale_factor, level);

  if (depth_proposal.size() == size)
  {
    LOG(INFO) << "Copy depth proposals " << depth_proposal.size() << " to level0 "
              << mu_->size();
    depth_proposal.copyTo(*mu_);
    depth_proposal_sigma2.copyTo(*sigma2_);
  }
  else
  {
    float downscale_factor = 0.5f*((float)size.width()/(float)depth_proposal.width()+
                                   (float)size.height()/(float)depth_proposal.height());

    if (params_->verbose>2)
      LOG(INFO) << "depth proposal downscaled to level: " << level << "; size: " << size
                << "; downscale_factor: " << downscale_factor;

    imp::cu::resample(*mu_, depth_proposal);
    imp::cu::resample(*sigma2_, depth_proposal_sigma2);
  }

  // assuming we receive the camera matrix for level0
  if  (level == 0)
  {
    cams_ = cams;
  }
  else
  {
    for (auto cam : cams)
    {
      cu::PinholeCamera scaled_cam = cam * scale_factor;
      cams_.push_back(scaled_cam);
    }
  }
}

//------------------------------------------------------------------------------
void SolverEpipolarStereoPrecondHuberL1::init()
{
  u_->setValue(0.0f);
  pu_->setValue(0.0f);
  q_->setValue(0.0f);
  // other variables are init and/or set when needed!
}

//------------------------------------------------------------------------------
void SolverEpipolarStereoPrecondHuberL1::init(const SolverStereoAbstract& rhs)
{
  const SolverEpipolarStereoPrecondHuberL1* from =
      dynamic_cast<const SolverEpipolarStereoPrecondHuberL1*>(&rhs);

  float inv_sf = 1./params_->ctf.scale_factor; // >1 for adapting prolongated disparities

  if(params_->ctf.apply_median_filter)
  {
    imp::cu::filterMedian3x3(*from->u0_, *from->u_);
    imp::cu::resample(*u_, *from->u0_, imp::InterpolationMode::point, false);
  }
  else
  {
    imp::cu::resample(*u_, *from->u_, imp::InterpolationMode::point, false);
  }
  *u_ *= inv_sf;

  imp::cu::resample(*pu_, *from->pu_, imp::InterpolationMode::point, false);
  imp::cu::resample(*q_, *from->q_, imp::InterpolationMode::point, false);
}

//------------------------------------------------------------------------------
void SolverEpipolarStereoPrecondHuberL1::solve(std::vector<ImagePtr> images)
{
  if (params_->verbose > 0)
    std::cout << "SolverEpipolarStereoPrecondHuberL1: solving level " << level_ << " with " << images.size() << " images" << std::endl;

  // sanity check:
  // TODO

  // constants
  constexpr float tau = 0.95f;
  constexpr float sigma = 0.95f;
  float lin_step = 0.5f;
  Fragmentation<16,16> frag(size_);
  constexpr float eta = 2.0f;

  // init
  u_->copyTo(*u_prev_);


  // check if a pointwise lambda is set in the parameters. otherwise we create
  // a local one to simplify kernel interfaces
  cu::ImageGpu32fC1::Ptr lambda;
  if (params_->lambda_pointwise)
    lambda = params_->lambda_pointwise;
  else
  {
    // make it as small as possible to reduce memory overhead. access is then
    // handled by the texture
    lambda.reset(new ImageGpu32fC1(1,1));
    lambda->setValue(params_->lambda);
  }
  lambda_tex_ = lambda->genTexture(false,hipFilterModePoint,
                                   hipAddressModeClamp, hipReadModeElementType);

  // textures
  i1_tex_ = images.at(0)->genTexture(false, hipFilterModeLinear);
  i2_tex_ = images.at(1)->genTexture(false, hipFilterModeLinear);
  u_tex_ = u_->genTexture(false, hipFilterModeLinear);
  u_prev_tex_ =  u_prev_->genTexture(false, hipFilterModeLinear);
  u0_tex_ =  u0_->genTexture(false, hipFilterModeLinear);
  pu_tex_ =  pu_->genTexture(false, hipFilterModeLinear);
  q_tex_ =  q_->genTexture(false, hipFilterModeLinear);
  ix_tex_ =  ix_->genTexture(false, hipFilterModeLinear);
  it_tex_ =  it_->genTexture(false, hipFilterModeLinear);
  xi_tex_ =  xi_->genTexture(false, hipFilterModeLinear);
  g_tex_ =  g_->genTexture(false, hipFilterModeLinear);
  mu_tex_ =  mu_->genTexture(false, hipFilterModeLinear);
  sigma2_tex_ =  sigma2_->genTexture(false, hipFilterModeLinear);
  pt_mu_tex_ =  pt_mu_->genTexture(false, hipFilterModeLinear);
  epi_vec_tex_ =  epi_vec_->genTexture(false, hipFilterModeLinear);



  // compute edge weight
  imp::cu::naturalEdges(*g_, *images.at(0),
                        params_->edge_sigma, params_->edge_alpha, params_->edge_q);



  // warping
  for (std::uint32_t warp = 0; warp < params_->ctf.warps; ++warp)
  {
    if (params_->verbose > 5)
      std::cout << "SOLVING warp iteration of Huber-L1 stereo model." << std::endl;

    u_->copyTo(*u0_);


    // compute warped spatial and temporal gradients
    k_warpedGradientsEpipolarConstraint
        <<<
          frag.dimGrid, frag.dimBlock
        >>> (ix_->data(), it_->data(), ix_->stride(),
             //todo mas
             pt_mu_->data(), epi_vec_->data(), pt_mu_->stride(),
             size_.width(), size_.height(),
             cams_.at(0), cams_.at(1), F_, T_mov_fix_,
             *i1_tex_, *i2_tex_, *u0_tex_,
             *mu_tex_);

    // compute preconditioner
    //! @todo  (MWE) fix preconditioner with pointwise lambda!
#if USE_EDGES
    k_preconditionerWeighted
        <<<
          frag.dimGrid, frag.dimBlock
        >>> (xi_->data(), xi_->stride(), xi_->width(), xi_->height(),
             params_->lambda, *ix_tex_, *g_tex_);
#else
    k_preconditioner
        <<<
          frag.dimGrid, frag.dimBlock
        >>> (xi_->data(), xi_->stride(), xi_->width(), xi_->height(),
             params_->lambda, *ix_tex_);
#endif

    for (std::uint32_t iter = 0; iter < params_->ctf.iters; ++iter)
    {
      // dual update kernel
      k_epiDualUpdate
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (pu_->data(), pu_->stride(), q_->data(), q_->stride(),
               size_.width(), size_.height(),
               epipoles_.at(0),
               params_->lambda, params_->eps_u, sigma, eta, //*lambda_tex_,
               *u_prev_tex_, *u0_tex_, *pu_tex_, *q_tex_, *ix_tex_, *it_tex_);

      // and primal update kernel
#if USE_EDGES
      // and primal update kernel
      k_primalUpdateWeighted
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (u_->data(), u_prev_->data(), u_->stride(),
               size_.width(), size_.height(),
               tau, lin_step, *lambda_tex_,
               *u_tex_, *u0_tex_, *pu_tex_, *q_tex_, *ix_tex_, *xi_tex_, *g_tex_);
#else
      k_epiPrimalUpdate
          <<<
            frag.dimGrid, frag.dimBlock
          >>> (u_->data(), u_prev_->data(), u_->stride(),
               size_.width(), size_.height(),
               epipoles_.at(0),
               params_->lambda, tau, lin_step, //*lambda_tex_,
               *u_tex_, *u0_tex_, *pu_tex_, *q_tex_, *ix_tex_, *xi_tex_);
#endif
    } // iters
    lin_step /= 1.2f;

  } // warps



  IMP_CUDA_CHECK();
}



} // namespace cu
} // namespace imp

